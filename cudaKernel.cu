#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hipfft/hipfft.h>

#define THREADS_PER_BLOCK 1024
#define PI 3.1415926535857932384626433f

#define FULL_MASK 0xffffffff

typedef float2 Complex;

Complex *h_data;
Complex *d_data;
Complex *d_dataCWT;
float   *d_noise_max;
float   *d_noise_min;

__device__ inline void floatAtomicAdd (float *address, float value) {
	int oldval, newval, readback;

	oldval = __float_as_int(*address);
	newval = __float_as_int(__int_as_float(oldval) + value);
	while ((readback=atomicCAS((int *)address, oldval, newval)) != oldval) {
		oldval = readback;
		newval = __float_as_int(__int_as_float(oldval) + value);
	}
}

__device__ inline void floatAtomicMin(float *address, float value) {
	int oldval, newval, readback;

	oldval = __float_as_int(*address);
	newval = __float_as_int(fminf(__int_as_float(oldval), value));
	while ((readback=atomicCAS((int *)address, oldval, newval)) != oldval) {
		oldval = readback;
		newval = __float_as_int(fminf(__int_as_float(oldval), value));
	}
}

__device__ inline void floatAtomicMax(float *address, float value) {
	int oldval, newval, readback;

	oldval = __float_as_int(*address);
	newval = __float_as_int(fmaxf(__int_as_float(oldval), value));
	while ((readback=atomicCAS((int *)address, oldval, newval)) != oldval) {
		oldval = readback;
		newval = __float_as_int(fmaxf(__int_as_float(oldval), value));
	}
}

static __global__ void complex_wfilith(Complex* dataCWT, Complex *data, int length, int voice) {
	//gloabl memory index
    int i = blockIdx.x;
    int j = blockIdx.y * THREADS_PER_BLOCK + threadIdx.x;

    float mu, cs, ks, w, a, waveletFFT;

    if (j < length) {
    	int index = i * length + j;

    	a = powf(2, (i + 1.f) / voice);
		mu = 2 * PI;
		cs = powf(1 + exp(-1.f * mu * mu) - 2 * expf(-0.75f * mu * mu), -0.5f) * powf(PI, -0.25f) * sqrtf(a) / sqrtf(2.f * PI);
		ks = expf(-0.5f * mu * mu);
		if (j <= length / 2) {
			w = j * a * 2. * PI / length;
		} else {
			w = (j - length) * a * 2.f * PI / length;
		}

		waveletFFT = cs * (expf(-0.5f * (mu - w) * (mu - w)) - ks * expf(-0.5f * w * w)) / length;

    	dataCWT[index].x = data[j].x * waveletFFT;
    	dataCWT[index].y = data[j].y * waveletFFT;
    }
}

static __global__ void noise_model(Complex* dataCWT, float* max, float *min, int ib, int ie, int length) {
	//gloabl memory index
    int i = blockIdx.x;
    int j = blockIdx.y * THREADS_PER_BLOCK + threadIdx.x;
	
	if (j == 0) {
		max[i] = 0;
		min[i] = 1e40;
	}

	__syncthreads();

	float maxd = 0;
	float mind = 1e40;
	if (j >= ib && j <= ie) {
    	int index = i * length + j;

    	maxd = sqrtf(dataCWT[index].x * dataCWT[index].x + dataCWT[index].y * dataCWT[index].y);
    	mind = maxd;
    }

    for (int i = 16; i; i >>= 1) {
		maxd = fmaxf(__shfl_down_sync(FULL_MASK, maxd, i), maxd);
		mind = fminf(__shfl_down_sync(FULL_MASK, mind, i), mind);
	}

	__shared__ float blockmaxd;
	__shared__ float blockmind;
	blockmaxd = 0;
	blockmind = 1e40;

	__syncthreads();

	if (threadIdx.x % 32 == 0) {
		floatAtomicMax(&blockmaxd, maxd);
		floatAtomicMin(&blockmind, mind);
	}

	__syncthreads();

	if (threadIdx.x == 0) {
		floatAtomicMax(max + i, blockmaxd);
		floatAtomicMin(min + i, blockmind);
	}
}

static __global__ void designal(Complex* dataCWT, int dataLength, float *max, float* min) {
	//gloabl memory index
    int i = blockIdx.x;
    int j = blockIdx.y * THREADS_PER_BLOCK + threadIdx.x;

    if (j < dataLength) {
    	int index = i * dataLength + j;

    	float noiseLevel = min[i] + (max[i] - min[i]) * 0.99;
    	float amp = sqrtf(dataCWT[index].x * dataCWT[index].x + dataCWT[index].y * dataCWT[index].y);

    	if (amp > noiseLevel) {
    		dataCWT[index].x *= noiseLevel / amp;
    		dataCWT[index].y *= noiseLevel / amp;
    	}
    }
}

static __global__ void denoise(Complex* dataCWT, int dataLength, float *max, float* min) {
	//gloabl memory index
    int i = blockIdx.x;
    int j = blockIdx.y * THREADS_PER_BLOCK + threadIdx.x;

    if (j < dataLength) {
    	int index = i * dataLength + j;

    	float noiseLevel = min[i] + (max[i] - min[i]) * 0.99;
    	float amp = sqrtf(dataCWT[index].x * dataCWT[index].x + dataCWT[index].y * dataCWT[index].y);

    	if (amp > noiseLevel) {
    		dataCWT[index].x *= (amp - noiseLevel) / amp;
    		dataCWT[index].y *= (amp - noiseLevel) / amp;
    	} else {
    		dataCWT[index].x = 0.f;
    		dataCWT[index].y = 0.f;
    	}
    }
}

static __global__ void complex_wfilith_inverse(Complex* dataCWT, int length, int voice) {
	//gloabl memory index
    int i = blockIdx.x;
    int j = blockIdx.y * THREADS_PER_BLOCK + threadIdx.x;

    float mu, cs, ks, w, a, waveletFFT;

    if (j < length) {
    	int index = i * length + j;

    	a = powf(2, (i + 1.f) / voice);
		mu = 2.f * PI;
		cs = powf(1.f + exp(-1.f * mu * mu) - 2 * expf(-0.75f * mu * mu), -0.5f) * powf(PI, -0.25f) * sqrtf(a) / sqrtf(2.f * PI);
		ks = expf(-0.5f * mu * mu);
		if (j <= length / 2) {
			w = j * a * 2. * PI / length;
		} else {
			w = (j - length) * a * 2.f * PI / length;
		}

		waveletFFT = cs * (expf(-0.5f * (mu - w) * (mu - w)) - ks * expf(-0.5f * w * w)) / length / a * logf(2.f) / (0.161252589430996f / 4.f / PI) / voice;

    	dataCWT[index].x *= waveletFFT;
    	dataCWT[index].y *= waveletFFT;
    }
}

static __global__ void inverse_cwt_sum(Complex* dataCWT, Complex* data, int length) {
	//gloabl memory index
    int i = blockIdx.x;
    int j = blockIdx.y * THREADS_PER_BLOCK + threadIdx.x;

    if (j < length) {
    	int index = i * length + j;
    	floatAtomicAdd(&data[j].x, dataCWT[index].x);
    }
}

extern "C"
void cuda_cwt_forward(double *data, int dataLength, int numOct, int numVoice) {
	hipfftHandle forwardPlan;
	hipfftHandle inversePlan;

	// initial the GPU memory
	if (hipMalloc((void**)&d_data,        sizeof(Complex) * dataLength)                     != hipSuccess ||
		hipMalloc((void**)&d_noise_max,   sizeof(float)   * numOct * numVoice)              != hipSuccess ||
		hipMalloc((void**)&d_noise_min,   sizeof(float)   * numOct * numVoice)              != hipSuccess ||
		hipMalloc((void**)&d_dataCWT,     sizeof(Complex) * numOct * numVoice * dataLength) != hipSuccess) {
		fprintf(stderr, "Error allocate memory for GPU parameters\n");
		exit(0);
	}

	h_data = (Complex *)malloc(sizeof(Complex) * dataLength);
	if (h_data == NULL) {
		fprintf(stderr, "Cannot allocate h_data\n");
		exit(0);
	}

	for (int i = 0; i < dataLength; i++) {
		h_data[i].x = data[i];
		h_data[i].y = 0;
	}

	if (hipMemcpy(d_data, h_data, sizeof(Complex) * dataLength, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "Cannot copy from CPU to GPU\n");
		exit(0);
	}

	hipfftPlan1d(&forwardPlan, dataLength, HIPFFT_C2C, 1);

	hipfftPlan1d(&inversePlan, dataLength, HIPFFT_C2C, numOct * numVoice);

	hipfftExecC2C(forwardPlan, (hipfftComplex *)d_data, (hipfftComplex *)d_data, HIPFFT_FORWARD);

	complex_wfilith<<<dim3(numOct * numVoice, dataLength/THREADS_PER_BLOCK + 1), THREADS_PER_BLOCK>>>(d_dataCWT, d_data, dataLength, numVoice);

	hipfftExecC2C(inversePlan, (hipfftComplex *)d_dataCWT, (hipfftComplex *)d_dataCWT, HIPFFT_BACKWARD);

	hipfftDestroy(forwardPlan);
	hipfftDestroy(inversePlan);

}

extern "C"
void cuda_initial_noise_model(int ib, int ie, int dataLength, int length) {
	noise_model<<<dim3(length, dataLength/THREADS_PER_BLOCK + 1), THREADS_PER_BLOCK>>>(d_dataCWT, d_noise_max, d_noise_min, ib, ie, dataLength);
}

extern "C"
void cuda_soft_thresholding(int dataLength, int length, int mode) {
	if (mode == 0) {
		designal<<<dim3(length, dataLength/THREADS_PER_BLOCK + 1), THREADS_PER_BLOCK>>>(d_dataCWT, dataLength, d_noise_max, d_noise_min);
	} else {
		denoise<<<dim3(length, dataLength/THREADS_PER_BLOCK + 1), THREADS_PER_BLOCK>>>(d_dataCWT, dataLength, d_noise_max, d_noise_min);
	}
}

extern "C"
void cuda_cwt_inverse(double *data, int dataLength, int numOct, int numVoice) {
	hipfftHandle plan;

	if (hipMemset(d_data, 0, sizeof(Complex) * dataLength) != hipSuccess) {
		fprintf(stderr, "Cannot set memory to zeros\n");
		exit(0);
	}

	hipfftPlan1d(&plan, dataLength, HIPFFT_C2C, numOct * numVoice);

	hipfftExecC2C(plan, (hipfftComplex *)d_dataCWT, (hipfftComplex *)d_dataCWT, HIPFFT_FORWARD);

	complex_wfilith_inverse<<<dim3(numOct * numVoice, dataLength/THREADS_PER_BLOCK + 1), THREADS_PER_BLOCK>>>(d_dataCWT, dataLength, numVoice);

	hipfftExecC2C(plan, (hipfftComplex *)d_dataCWT, (hipfftComplex *)d_dataCWT, HIPFFT_BACKWARD);

	inverse_cwt_sum<<<dim3(numOct * numVoice, dataLength/THREADS_PER_BLOCK + 1), THREADS_PER_BLOCK>>>(d_dataCWT, d_data, dataLength);

	if (hipMemcpy(h_data, d_data, sizeof(Complex) * dataLength, hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "Cannot copy from GPU to CPU\n");
		exit(0);
	}

	for (int i = 0; i < dataLength; i++) {
		data[i] = h_data[i].x;
	}

	hipfftDestroy(plan);

	free(h_data);

	hipFree(d_data);
	hipFree(d_dataCWT);
	hipFree(d_noise_max);
	hipFree(d_noise_min);
}